#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2022 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "cukd/builder.h"
// fcp = "find closest point" query
#include "cukd/fcp.h"

float2 *generatePoints(int N)
{
  std::cout << "generating " << N <<  " points" << std::endl;
  float2 *d_points = 0;
  CUKD_CUDA_CALL(MallocManaged((void**)&d_points,N*sizeof(float2)));
  for (int i=0;i<N;i++) {
    d_points[i].x = (float)drand48();
    d_points[i].y = (float)drand48();
  }
  return d_points;
}

__global__ void d_fcp(int *d_results,
                    float2 *d_queries,
                    int numQueries,
                    float2 *d_nodes,
                    int numNodes)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numQueries) return;

  d_results[tid]
    = cukd::fcp
    <cukd::TrivialFloatPointTraits<float2>>
    (d_queries[tid],d_nodes,numNodes);
}

void fcp(int *d_results,
         float2 *d_queries,
         int numQueries,
         float2 *d_nodes,
         int numNodes)
{
  int bs = 128;
  int nb = cukd::common::divRoundUp(numQueries,bs);
  d_fcp<<<nb,bs>>>(d_results,d_queries,numQueries,d_nodes,numNodes);
}

bool noneBelow(float2 *d_points, int N, int curr, int dim, float value)
{
  if (curr >= N) return true;
  return
    ((&d_points[curr].x)[dim] >= value)
    && noneBelow(d_points,N,2*curr+1,dim,value)
    && noneBelow(d_points,N,2*curr+2,dim,value);
}

bool noneAbove(float2 *d_points, int N, int curr, int dim, float value)
{
  if (curr >= N) return true;
  return
    ((&d_points[curr].x)[dim] <= value)
    && noneAbove(d_points,N,2*curr+1,dim,value)
    && noneAbove(d_points,N,2*curr+2,dim,value);
}

bool checkTree(float2 *d_points, int N, int curr=0)
{
  if (curr >= N) return true;

  int dim = cukd::BinaryTree::levelOf(curr)%4;
  float value = (&d_points[curr].x)[dim];
  
  if (!noneAbove(d_points,N,2*curr+1,dim,value))
    return false;
  if (!noneBelow(d_points,N,2*curr+2,dim,value))
    return false;
  
  return
    checkTree(d_points,N,2*curr+1)
    &&
    checkTree(d_points,N,2*curr+2);
}

int main(int ac, const char **av)
{
  using namespace cukd::common;

  int nPoints = 173;
  bool verify = false;
  // float maxQueryRadius = std::numeric_limits<float>::infinity();
  int nRepeats = 1;
  size_t nQueries = 10000000;
  for (int i=1;i<ac;i++) {
    std::string arg = av[i];
    if (arg[0] != '-')
      nPoints = std::stoi(arg);
    else if (arg == "-v")
      verify = true;
    else if (arg == "-nq")
      nQueries = atoi(av[++i]);
    else if (arg == "-nr")
      nRepeats = atoi(av[++i]);
    else
      throw std::runtime_error("known cmdline arg "+arg);
  }
  
  float2 *d_points = loadPoints<float2>("data_points",nPoints);//generatePoints(nPoints);
  // float2 *d_points = generatePoints(nPoints);
  
  {
    double t0 = getCurrentTime();
    std::cout << "calling builder..." << std::endl;
    cukd::buildTree<cukd::TrivialFloatPointTraits<float2>>(d_points,nPoints);
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done building tree, took " << prettyDouble(t1-t0) << "s" << std::endl;
  }

  if (verify) {
    std::cout << "checking tree..." << std::endl;
    if (!checkTree(d_points,nPoints))
      throw std::runtime_error("not a valid kd-tree!?");
    else
      std::cout << "... passed" << std::endl;
  }

  // float2 *d_queries = generatePoints(nQueries);
  float2 *d_queries = loadPoints<float2>("query_points",nQueries);
  int    *d_results;
  CUKD_CUDA_CALL(MallocManaged((void**)&d_results,nQueries*sizeof(int)));
  {
    double t0 = getCurrentTime();
    for (int i=0;i<nRepeats;i++) {
      fcp(d_results,d_queries,nQueries,d_points,nPoints);
    }
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done " << nRepeats << " iterations of 10M fcp queries, took " << prettyDouble(t1-t0) << "s" << std::endl;
    std::cout << "that is " << prettyDouble(nQueries*nRepeats/(t1-t0)) << " queries/s" << std::endl;
  }
  
  if (verify) {
    std::cout << "verifying ..." << std::endl;
    for (int i=0;i<nQueries;i++) {
      if (d_results[i] == -1) continue;
      
      float2 qp = d_queries[i];
      float reportedDist
        = cukd::distance
        <cukd::TrivialFloatPointTraits<float2>>
        (qp,d_points[d_results[i]]);
      // float reportedDist = cukd::distance<cukd::TrivialFloatPointTraits<float2>>(qp,d_points[d_results[i]]);
      for (int j=0;j<nPoints;j++) {
        float dist_j = cukd::distance
          <cukd::TrivialFloatPointTraits<float2>>
          (qp,d_points[j]);
        // float dist_j = cukd::distance(qp,d_points[j]);
        if (dist_j < reportedDist) {
          printf("for query %i: found offending point %i (%f %f) with dist %f (vs %f)\n",
                 i,
                 j,
                 d_points[j].x,
                 d_points[j].y,
                 dist_j,
                 reportedDist);
          
          throw std::runtime_error("verification failed ...");
        }
      }
    }
    std::cout << "verification succeeded... done." << std::endl;
  }
}
