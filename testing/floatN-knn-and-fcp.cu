#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2023 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "cukd/builder.h"
// fcp = "find closest point" query
#include "cukd/fcp.h"
#include "cukd/knn.h"
#include <queue>
#include <iomanip>

#if D_FROM_CMAKE == 2
using floatN = float2;
#elif D_FROM_CMAKE == 3
using floatN = float3;
#elif D_FROM_CMAKE == 4
using floatN = float4;
#elif D_FROM_CMAKE == 8
using floatN = cukd::vec_float<8>;
#else
#pragma error("error ... should get a value of 2, 3, or 4 from cmakefile...")
#endif

using namespace cukd;


#if EXPLICIT_DIM
struct PointAndDim {
  floatN point;
  int    dim;
};

struct PointAndDim_traits : public cukd::default_node_traits<floatN> {
  enum { has_explicit_dim = true };
  using node_t = PointAndDim;
  
  static inline __both__ const point_t &get_point(const node_t &n) { return n.point; }
  static inline __both__ float get_coord(const PointAndDim &n, int d)
  { return cukd::get_coord(n.point,d); }
  
  static inline __both__ int  get_dim(const PointAndDim &p) 
  { return p.dim; }
	   
  static inline __both__ void set_dim(PointAndDim &p, int dim) 
  { p.dim = dim; }
};

using node_t = PointAndDim;
using node_traits = PointAndDim_traits;
#else
using node_t = floatN;
using node_traits = default_node_traits<floatN>;
#endif



__global__
void d_fcp(unsigned long long *d_stats,
           float   *d_results,
           floatN  *d_queries,
           int      numQueries,
#if CUKD_IMPROVED_TRAVERSAL
           const cukd::box_t<floatN> *d_bounds,
#endif
           node_t  *d_nodes,
           int      numNodes,
           float    cutOffRadius)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numQueries) return;

  using point_t = floatN;
  point_t queryPos = d_queries[tid];
  FcpSearchParams params;
  params.cutOffRadius = cutOffRadius;
  int closestID
    = cukd::fcp
    <node_t,node_traits>
    (d_stats,queryPos,
#if CUKD_IMPROVED_TRAVERSAL
     *d_bounds,
#endif
     d_nodes,numNodes,params);
  
  d_results[tid]
    = (closestID < 0)
    ? INFINITY
    : distance(queryPos,node_traits::get_point(d_nodes[closestID]));
}




template<typename CandidateList>
__global__
void d_knn(unsigned long long *d_stats,
           float   *d_results,
           floatN  *d_queries,
           int      numQueries,
#if CUKD_IMPROVED_TRAVERSAL
           const cukd::box_t<floatN> *d_bounds,
#endif
           node_t  *d_nodes,
           int      numNodes,
           float    cutOffRadius)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numQueries) return;
  
  CandidateList result(cutOffRadius);
  float sqrDist
    = cukd::knn(d_stats,result,d_queries[tid],
#if CUKD_IMPROVED_TRAVERSAL
                *d_bounds,
#endif
                d_nodes,numNodes);
  d_results[tid] = sqrtf(sqrDist);
}



void run_kernel(float  *d_results,
                floatN *d_queries,
                int     numQueries,
#if CUKD_IMPROVED_TRAVERSAL
                const cukd::box_t<floatN> *d_bounds,
#endif
                node_t *d_nodes,
                int     numNodes,
#if USE_KNN
                int k,
#endif
                float   cutOffRadius
                )
{
  int bs = 128;
  int nb = divRoundUp(numQueries,bs);
  unsigned long long *d_stats = 0;
  static bool firstTime = true;
  if (firstTime) {
    hipMallocManaged((char **)&d_stats,sizeof(*d_stats));
    *d_stats = 0;
  }
#if USE_KNN
  if (k == 4)
    d_knn<FixedCandidateList<4>><<<nb,bs>>>
      (d_stats,d_results,d_queries,numQueries,
# if CUKD_IMPROVED_TRAVERSAL
       d_bounds,
# endif
       d_nodes,numNodes,cutOffRadius);
  else if (k == 8)
    d_knn<FixedCandidateList<8>><<<nb,bs>>>
      (d_stats,d_results,d_queries,numQueries,
# if CUKD_IMPROVED_TRAVERSAL
       d_bounds,
# endif
       d_nodes,numNodes,cutOffRadius);
  else if (k == 64)
    d_knn<HeapCandidateList<64>><<<nb,bs>>>
      (d_stats,d_results,d_queries,numQueries,
# if CUKD_IMPROVED_TRAVERSAL
       d_bounds,
# endif
       d_nodes,numNodes,cutOffRadius);
  else if (k == 20)
    d_knn<HeapCandidateList<20>><<<nb,bs>>>
      (d_stats,d_results,d_queries,numQueries,
# if CUKD_IMPROVED_TRAVERSAL
       d_bounds,
# endif
       d_nodes,numNodes,cutOffRadius);
  else if (k == 50)
    d_knn<HeapCandidateList<50>><<<nb,bs>>>
      (d_stats,d_results,d_queries,numQueries,
# if CUKD_IMPROVED_TRAVERSAL
       d_bounds,
# endif
       d_nodes,numNodes,cutOffRadius);
  else
    throw std::runtime_error("unsupported k for knn queries");
#else
  d_fcp<<<nb,bs>>>
    (d_stats,d_results,d_queries,numQueries,
# if CUKD_IMPROVED_TRAVERSAL
     d_bounds,
# endif
     d_nodes,numNodes,cutOffRadius);
#endif
  if (firstTime) {
    hipDeviceSynchronize();
    std::cout << "KDTREE_STATS " << *d_stats << std::endl;
    hipFree(d_stats);
    firstTime = false;
  }
}

#if EXPLICIT_DIM
__global__ void copyPoints(PointAndDim *d_points,
                           floatN *d_inputs,
                           int numPoints)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numPoints) return;
  d_points[tid].point = d_inputs[tid];
}
#endif

template<typename node_t, typename node_traits>
void verifyKNN(int pointID,
               int k,
               float maxRadius,
               floatN *points, int numPoints,
               floatN queryPoint,
               float reportedResult)
{
  using point_t = typename node_traits::point_t;
  std::priority_queue<float> closest_k;
  for (int i=0;i<numPoints;i++) {
    point_t point_i = node_traits::get_point(points[i]);
    float d = sqrDistance(queryPoint,point_i);
    if (d >= maxRadius*maxRadius)
      continue;
    
    closest_k.push(d);
    if (closest_k.size() > k)
      closest_k.pop();
  }
  
  float actualResult = (closest_k.size() == k) ? sqrtf(closest_k.top()) : maxRadius;
  
  // check if the top 21-ish bits are the same; this will allow the
  // compiler to produce slightly different results on host and device
  // (usually caused by it uses madd's on one and separate +/* on
  // t'other...
  bool closeEnough
    =  /* this catches result==inf:*/
    (actualResult == reportedResult)
    || /* this catches bit errors: */
    (fabsf(actualResult - reportedResult)/std::max(actualResult,reportedResult) <= 1e-6f);
  
  if (!closeEnough) {
    std::cout << "for point #" << pointID << ": "
              << "verify found max dist " << std::setprecision(10) << actualResult
              << " (bits " << (int*)(uint64_t)((uint32_t&)actualResult)
              << "), knn reported " << reportedResult
              << " (bits " << (int*)(uint64_t)((uint32_t&)reportedResult)
              << "), difference is " << (actualResult-reportedResult)
              << std::endl;
    throw std::runtime_error("verification failed");
  }
}


template<typename node_t, typename node_traits>
void verifyFCP(int pointID,
               float cutOffRadius,
               node_t *points, int numPoints,
               floatN queryPoint,
               float reportedResult)
{
  using point_t = typename node_traits::point_t;
  float actualResult = INFINITY;
  for (int i=0;i<numPoints;i++) {
    point_t point_i = node_traits::get_point(points[i]);
    float d = sqrDistance(queryPoint,point_i);
    if (d >= cutOffRadius*cutOffRadius)
      continue;

    actualResult = std::min(actualResult,sqrtf(d));
  }
  
  
  // check if the top 21-ish bits are the same; this will allow the
  // compiler to produce slightly different results on host and device
  // (usually caused by it uses madd's on one and separate +/* on
  // t'other...
  bool closeEnough
    =  /* this catches result==inf:*/
    (actualResult == reportedResult)
    || /* this catches bit errors: */
    (fabsf(actualResult - reportedResult)/std::max(actualResult,reportedResult) <= 1e-6f);
  
  if (!closeEnough) {
    std::cout << "for point #" << pointID << ": "
              << "verify found max dist " << std::setprecision(10) << actualResult
              << " (bits " << (int*)(uint64_t)((uint32_t&)actualResult)
              << "), knn reported " << reportedResult
              << " (bits " << (int*)(uint64_t)((uint32_t&)reportedResult)
              << "), difference is " << (actualResult-reportedResult)
              << std::endl;
    throw std::runtime_error("verification failed");
  }
}


int main(int ac, const char **av)
{
  using namespace cukd::common;

  int    numPoints = 173;
  bool   verify = false;
  int    nRepeats = 1;
  size_t numQueries = 10000000;
  float  cutOffRadius = std::numeric_limits<float>::infinity();
#if USE_KNN
  int    k = 50;
#endif
  for (int i=1;i<ac;i++) {
    std::string arg = av[i];
    if (arg[0] != '-')
      numPoints = std::stoi(arg);
    else if (arg == "-v")
      verify = true;
    else if (arg == "-nq")
      numQueries = atoi(av[++i]);
    else if (arg == "-nr")
      nRepeats = atoi(av[++i]);
    else if (arg == "-r")
      cutOffRadius = std::stof(av[++i]);
#if USE_KNN
    else if (arg == "-k")
      k = std::stoi(av[++i]);
#endif
    else
      throw std::runtime_error("known cmdline arg "+arg);
  }
  
  floatN *d_inputs = loadPoints<floatN>("data_points",numPoints);
#if EXPLICIT_DIM
  PointAndDim *d_points;
  hipMallocManaged((void**)&d_points,numPoints*sizeof(*d_points));
  copyPoints<<<divRoundUp(numPoints,128),128>>>
    (d_points,d_inputs,numPoints);
  using node_t = PointAndDim;
#else
  floatN *d_points = d_inputs;
  using node_t = floatN;
#endif
  
#if CUKD_IMPROVED_TRAVERSAL
  cukd::box_t<floatN> *d_bounds;
  hipMalloc((void**)&d_bounds,sizeof(cukd::box_t<floatN>));
  cukd::computeBounds<node_t,node_traits>
    (d_bounds,d_points,numPoints);
#endif
  {
    double t0 = getCurrentTime();
    std::cout << "calling builder..." << std::endl;
    cukd::buildTree<node_t,node_traits>
      (d_points,numPoints);
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done building tree, took "
              << prettyDouble(t1-t0) << "s" << std::endl;
  }
  
  floatN *d_queries = loadPoints<floatN>("query_points",numQueries);
  float  *d_results;
  CUKD_CUDA_CALL(MallocManaged((void**)&d_results,numQueries*sizeof(*d_results)));
  {
    double t0 = getCurrentTime();
    for (int i=0;i<nRepeats;i++) {
      run_kernel
        (d_results,d_queries,numQueries,
#if CUKD_IMPROVED_TRAVERSAL
         d_bounds,
#endif
         d_points,numPoints,
#if USE_KNN
         k,
#endif
         cutOffRadius);
    }
    CUKD_CUDA_SYNC_CHECK();
    double t1 = getCurrentTime();
    std::cout << "done " << nRepeats
              << " iterations of " << numQueries
              << " fcp queries, took " << prettyDouble(t1-t0)
              << "s" << std::endl;
    std::cout << "that is " << prettyDouble(numQueries*nRepeats/(t1-t0))
              << " queries/s" << std::endl;
  }
  
  if (verify) {
    std::cout << "verifying ..." << std::endl;
    for (int i=0;i<numQueries;i++) {
      floatN qp           = d_queries[i];
      float  reportedResult = d_results[i];
#if USE_KNN
      verifyKNN<node_t,node_traits>
        (i,k,cutOffRadius,d_points,numPoints,qp,reportedResult);
#else
      verifyFCP<node_t,node_traits>
        (i,cutOffRadius,d_points,numPoints,qp,reportedResult);
#endif          
    }
  }
  std::cout << "verification succeeded... done." << std::endl;
}
  
